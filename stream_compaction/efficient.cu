#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define BLOCK_SIZE 256

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpsweep(int d, int n, int* data) {
            int k = blockIdx.x * blockDim.x + threadIdx.x;

            int stride = 1 << (d + 1);
            int numWorkItems = n / stride;

            if (k >= numWorkItems) return;

            int right = (k + 1) * stride - 1;
            int left = right - (1 << d);

            data[right] += data[left];
        }

        __global__ void kernDownsweep(int d, int n, int* data) {
            int k = blockIdx.x * blockDim.x + threadIdx.x;

            int stride = 1 << (d + 1);
            int numWorkItems = n / stride;

            if (k >= numWorkItems) return;

            int right = (k + 1) * stride - 1;
            int left = right - (1 << d);

            int t = data[left];
            data[left] = data[right];
            data[right] = t + data[right];
        }

        __global__ void kernExtractBit(int n, int bit, int* idata, int* odata) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;
            
            unsigned int ukey = (static_cast<unsigned int>(idata[idx]) ^ 0x80000000u);
            int b = (ukey >> bit) & 1u;
            odata[idx] = 1 - b;
        }

        __global__ void kernScatterByBit(int n, int* idata, int* bitArray,
            int* scanned, int* odata, int totalFalses) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;

            int bit = bitArray[idx];
            if (bit == 1) {
                odata[scanned[idx]] = idata[idx];
            }
            else {
                int pos = totalFalses + (idx - scanned[idx]);
                odata[pos] = idata[idx];
            }
        }

        void scanDevice(int n, int* dev_out, const int* dev_in) {
            int log2n = ilog2ceil(n);
            int m = 1 << log2n;

            int* dev_buf;
            hipMalloc(&dev_buf, m * sizeof(int));

            hipMemcpy(dev_buf, dev_in, n * sizeof(int), hipMemcpyDeviceToDevice);

            if (m > n) {
                hipMemset(dev_buf + n, 0, (m - n) * sizeof(int));
            }

            for (int d = 0; d < log2n; d++) {
                int numWorkItems = m >> (d + 1);
                int blocks = (numWorkItems + BLOCK_SIZE - 1) / BLOCK_SIZE;
                kernUpsweep<<<blocks, BLOCK_SIZE >>>(d, m, dev_buf);
            }

            hipMemset(dev_buf + (m - 1), 0, sizeof(int));

            for (int d = log2n - 1; d >= 0; d--) {
                int numWorkItems = m >> (d + 1);
                int blocks = (numWorkItems + BLOCK_SIZE - 1) / BLOCK_SIZE;
                kernDownsweep<<<blocks, BLOCK_SIZE >>>(d, m, dev_buf);
            }

            hipMemcpy(dev_out, dev_buf, n * sizeof(int), hipMemcpyDeviceToDevice);

            hipFree(dev_buf);
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            if (n <= 0) {
                return;
            }
            int* dev_in, * dev_out;
            hipMalloc(&dev_in, n * sizeof(int));
            hipMalloc(&dev_out, n * sizeof(int));
            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            scanDevice(n, dev_out, dev_in);
            timer().endGpuTimer();

            hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_in);
            hipFree(dev_out);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int* dev_in, * dev_flags, * dev_indices, * dev_out;

            hipMalloc(&dev_in, n * sizeof(int));
            hipMalloc(&dev_flags, n * sizeof(int));
            hipMalloc(&dev_indices, n * sizeof(int));
            hipMalloc(&dev_out, n * sizeof(int));

            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

            timer().startGpuTimer();

            timer().startGpuSubTimer("map");
            StreamCompaction::Common::kernMapToBoolean<<<gridSize, BLOCK_SIZE >>>(n, dev_flags, dev_in);
            hipDeviceSynchronize();
            timer().endGpuSubTimer();

            timer().startGpuSubTimer("scan");
            scanDevice(n, dev_indices, dev_flags);
            hipDeviceSynchronize();
            timer().endGpuSubTimer();

            int lastScan, lastFlag;
            hipMemcpy(&lastScan, dev_indices + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&lastFlag, dev_flags + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
            int validCount = lastScan + lastFlag;

            timer().startGpuSubTimer("scatter");
            StreamCompaction::Common::kernScatter<<<gridSize, BLOCK_SIZE >>>(n, dev_out, dev_in, dev_flags, dev_indices);
            hipDeviceSynchronize();
            timer().endGpuSubTimer();

            timer().endGpuTimer();

            if (validCount > 0) {
                hipMemcpy(odata, dev_out, validCount * sizeof(int), hipMemcpyDeviceToHost);
            }

            hipFree(dev_in);
            hipFree(dev_flags);
            hipFree(dev_indices);
            hipFree(dev_out);

            return validCount;
        }

        void radixSort(int n, int* odata, const int* idata) {
            int* dev_in, * dev_out, * dev_bits, * dev_scanned;
            hipMalloc(&dev_in, n * sizeof(int));
            hipMalloc(&dev_out, n * sizeof(int));
            hipMalloc(&dev_bits, n * sizeof(int));
            hipMalloc(&dev_scanned, n * sizeof(int));

            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

            timer().startGpuTimer();

            for (int bit = 0; bit < 32; bit++) {
                kernExtractBit<<<gridSize, BLOCK_SIZE >>>(n, bit, dev_in, dev_bits);

                scanDevice(n, dev_scanned, dev_bits);

                int lastScan = 0, lastIsZero = 0;
                hipMemcpy(&lastScan, dev_scanned + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&lastIsZero, dev_bits + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
                const int totalZeros = lastScan + lastIsZero;

                kernScatterByBit<<<gridSize, BLOCK_SIZE >>>(n, dev_in, dev_bits, dev_scanned, dev_out, totalZeros);

                std::swap(dev_in, dev_out);
            }

            timer().endGpuTimer();

            hipMemcpy(odata, dev_in, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_in);
            hipFree(dev_out);
            hipFree(dev_bits);
            hipFree(dev_scanned);
        }
    }
}
