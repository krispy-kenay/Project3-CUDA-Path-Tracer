#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#include "../src/sceneStructs.h"

#define BLOCK_SIZE 256

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpsweep(int d, int n, int* data) {
            int k = blockIdx.x * blockDim.x + threadIdx.x;

            int stride = 1 << (d + 1);
            int numWorkItems = n / stride;

            if (k >= numWorkItems) return;

            int right = (k + 1) * stride - 1;
            int left = right - (1 << d);

            data[right] += data[left];
        }

        __global__ void kernDownsweep(int d, int n, int* data) {
            int k = blockIdx.x * blockDim.x + threadIdx.x;

            int stride = 1 << (d + 1);
            int numWorkItems = n / stride;

            if (k >= numWorkItems) return;

            int right = (k + 1) * stride - 1;
            int left = right - (1 << d);

            int t = data[left];
            data[left] = data[right];
            data[right] = t + data[right];
        }

        __global__ void kernExtractBit(int n, int bit, int* idata, int* odata) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;
            
            unsigned int ukey = (static_cast<unsigned int>(idata[idx]) ^ 0x80000000u);
            int b = (ukey >> bit) & 1u;
            odata[idx] = 1 - b;
        }

        __global__ void kernScatterByBit(int n,
            int* bitArray,
            int* scanned,
            PathSegment* pathsIn, PathSegment* pathsOut,
            ShadeableIntersection* isectsIn, ShadeableIntersection* isectsOut,
            int totalFalses)
        {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;

            int bit = bitArray[idx];
            int newPos;
            if (bit == 1) {
                newPos = scanned[idx];
            }
            else {
                newPos = totalFalses + (idx - scanned[idx]);
            }

            pathsOut[newPos] = pathsIn[idx];
            isectsOut[newPos] = isectsIn[idx];
        }

        __global__ void kernScatterPartition(int n, PathSegment* inPaths, PathSegment* outPaths, int* flags, int* scanned, int totalActives)
        {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            if (i >= n) return;

            int f = flags[i];
            int pos = f ? scanned[i] : (totalActives + (i - scanned[i]));

            outPaths[pos] = inPaths[i];
        }



        void scanDevice(int n, int* dev_out, const int* dev_in) {
            int log2n = ilog2ceil(n);
            int m = 1 << log2n;

            int* dev_buf;
            hipMalloc(&dev_buf, m * sizeof(int));

            hipMemcpy(dev_buf, dev_in, n * sizeof(int), hipMemcpyDeviceToDevice);

            if (m > n) {
                hipMemset(dev_buf + n, 0, (m - n) * sizeof(int));
            }

            for (int d = 0; d < log2n; d++) {
                int numWorkItems = m >> (d + 1);
                int blocks = (numWorkItems + BLOCK_SIZE - 1) / BLOCK_SIZE;
                kernUpsweep<<<blocks, BLOCK_SIZE >>>(d, m, dev_buf);
            }

            hipMemset(dev_buf + (m - 1), 0, sizeof(int));

            for (int d = log2n - 1; d >= 0; d--) {
                int numWorkItems = m >> (d + 1);
                int blocks = (numWorkItems + BLOCK_SIZE - 1) / BLOCK_SIZE;
                kernDownsweep<<<blocks, BLOCK_SIZE >>>(d, m, dev_buf);
            }

            hipMemcpy(dev_out, dev_buf, n * sizeof(int), hipMemcpyDeviceToDevice);

            hipFree(dev_buf);
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            if (n <= 0) {
                return;
            }
            int* dev_in, * dev_out;
            hipMalloc(&dev_in, n * sizeof(int));
            hipMalloc(&dev_out, n * sizeof(int));
            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            scanDevice(n, dev_out, dev_in);
            timer().endGpuTimer();

            hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_in);
            hipFree(dev_out);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, PathSegment* dev_in, PathSegment* dev_out) {
            int* dev_flags, * dev_indices;

            hipMalloc(&dev_flags, n * sizeof(int));
            hipMalloc(&dev_indices, n * sizeof(int));

            int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

            //timer().startGpuTimer();

            StreamCompaction::Common::kernMapToBoolean<<<gridSize, BLOCK_SIZE >>>(n, dev_flags, dev_in);
            hipDeviceSynchronize();

            scanDevice(n, dev_indices, dev_flags);
            hipDeviceSynchronize();

            int lastScan, lastFlag;
            hipMemcpy(&lastScan, dev_indices + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&lastFlag, dev_flags + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
            int validCount = lastScan + lastFlag;

            kernScatterPartition<<<gridSize, BLOCK_SIZE >>>(n, dev_in, dev_out, dev_flags, dev_indices, validCount);
            hipDeviceSynchronize();
            //timer().endGpuTimer();

            hipFree(dev_flags);
            hipFree(dev_indices);

            return validCount;
        }

        void radixSort(int n, PathSegment* dev_paths, PathSegment* dev_paths_tmp, ShadeableIntersection* dev_isects, ShadeableIntersection* dev_isects_tmp) {
            int* dev_bits, * dev_scanned;
            hipMalloc(&dev_bits, n * sizeof(int));
            hipMalloc(&dev_scanned, n * sizeof(int));

            int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

            //timer().startGpuTimer();

            for (int bit = 0; bit < 32; bit++) {
                kernExtractBit<<<gridSize, BLOCK_SIZE >>>(n, bit, (int*)(&(dev_isects->materialId)), dev_bits);

                scanDevice(n, dev_scanned, dev_bits);

                int lastScan = 0, lastIsZero = 0;
                hipMemcpy(&lastScan, dev_scanned + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&lastIsZero, dev_bits + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
                const int totalZeros = lastScan + lastIsZero;

                kernScatterByBit<<<gridSize, BLOCK_SIZE >>>(n, dev_bits, dev_scanned, dev_paths, dev_paths_tmp, dev_isects, dev_isects_tmp, totalZeros);

                std::swap(dev_paths, dev_paths_tmp);
                std::swap(dev_isects, dev_isects_tmp);
            }

            //timer().endGpuTimer();
            hipFree(dev_bits);
            hipFree(dev_scanned);
        }
    }
}
